#include "hip/hip_runtime.h"
/*
  mallocMC: Memory Allocator for Many Core Architectures.
  https://www.hzdr.de/crp

  Copyright 2014 Institute of Radiation Physics,
                 Helmholtz-Zentrum Dresden - Rossendorf

  Author(s):  Carlchristian Eckert - c.eckert ( at ) hzdr.de

  Permission is hereby granted, free of charge, to any person obtaining a copy
  of this software and associated documentation files (the "Software"), to deal
  in the Software without restriction, including without limitation the rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included in
  all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
  THE SOFTWARE.
*/

#include <iostream>
#include <assert.h>
#include <vector>
#include <numeric>
#include <stdio.h>

#include <hip/hip_runtime.h>


///////////////////////////////////////////////////////////////////////////////
// includes for mallocMC
///////////////////////////////////////////////////////////////////////////////
#include <mallocMC/mallocMC_hostclass.hpp>
#include <mallocMC/CreationPolicies.hpp>
#include <mallocMC/DistributionPolicies.hpp>
#include <mallocMC/OOMPolicies.hpp>
#include <mallocMC/ReservePoolPolicies.hpp>
#include <mallocMC/AlignmentPolicies.hpp>

///////////////////////////////////////////////////////////////////////////////
// Configuration for mallocMC
///////////////////////////////////////////////////////////////////////////////

// configurate the CreationPolicy "Scatter"
struct ScatterConfig{
    static constexpr auto pagesize = 4096;
    static constexpr auto accessblocks = 8;
    static constexpr auto regionsize = 16;
    static constexpr auto wastefactor = 2;
    static constexpr auto resetfreedpages = false;
};

struct ScatterHashParams{
    static constexpr auto hashingK = 38183;
    static constexpr auto hashingDistMP = 17497;
    static constexpr auto hashingDistWP = 1;
    static constexpr auto hashingDistWPRel = 1;
};


// configure the AlignmentPolicy "Shrink"
struct AlignmentConfig{
    static constexpr auto dataAlignment = 16;
};

// Define a new mMCator and call it ScatterAllocator
// which resembles the behaviour of ScatterAlloc
using ScatterAllocator = mallocMC::Allocator<
    mallocMC::CreationPolicies::Scatter<ScatterConfig, ScatterHashParams>,
    mallocMC::DistributionPolicies::Noop,
    mallocMC::OOMPolicies::ReturnNull,
    mallocMC::ReservePoolPolicies::SimpleCudaMalloc,
    mallocMC::AlignmentPolicies::Shrink<AlignmentConfig>
>;

///////////////////////////////////////////////////////////////////////////////
// End of mallocMC configuration
///////////////////////////////////////////////////////////////////////////////


__device__ int* arA;


__global__ void exampleKernel(ScatterAllocator::AllocatorHandle mMC){
    unsigned x = 42;
    if(threadIdx.x==0)
        arA = (int*) mMC.malloc(sizeof(int) * 32);

    x = mMC.getAvailableSlots(1);
    __syncthreads();
    arA[threadIdx.x] = threadIdx.x;
    printf("tid: %d array: %d slots %d\n", threadIdx.x, arA[threadIdx.x],x);

    if(threadIdx.x == 0)
        mMC.free(arA);
}


int main()
{
    ScatterAllocator mMC(1U*1024U*1024U*1024U); //1GB for device-side malloc

    exampleKernel<<<1,32>>>( mMC );
    std::cout << "Slots from Host: " << mMC.getAvailableSlots(1) << std::endl;

    return 0;
}
