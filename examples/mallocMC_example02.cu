#include "hip/hip_runtime.h"
/*
  mallocMC: Memory Allocator for Many Core Architectures.
  https://www.hzdr.de/crp

  Copyright 2014 Institute of Radiation Physics,
                 Helmholtz-Zentrum Dresden - Rossendorf

  Author(s):  Carlchristian Eckert - c.eckert ( at ) hzdr.de

  Permission is hereby granted, free of charge, to any person obtaining a copy
  of this software and associated documentation files (the "Software"), to deal
  in the Software without restriction, including without limitation the rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included in
  all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
  THE SOFTWARE.
*/

#include <cassert>
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <vector>

///////////////////////////////////////////////////////////////////////////////
// includes for mallocMC
///////////////////////////////////////////////////////////////////////////////
// basic files for mallocMC
#include <mallocMC/mallocMC_hostclass.hpp>

// Load all available policies for mallocMC
#include <mallocMC/AlignmentPolicies.hpp>
#include <mallocMC/CreationPolicies.hpp>
#include <mallocMC/DistributionPolicies.hpp>
#include <mallocMC/OOMPolicies.hpp>
#include <mallocMC/ReservePoolPolicies.hpp>

///////////////////////////////////////////////////////////////////////////////
// Configuration for mallocMC
///////////////////////////////////////////////////////////////////////////////

// configurate the CreationPolicy "Scatter"
struct ScatterConfig
{
    static constexpr auto pagesize = 4096;
    static constexpr auto accessblocks = 8;
    static constexpr auto regionsize = 16;
    static constexpr auto wastefactor = 2;
    static constexpr auto resetfreedpages = false;
};

struct ScatterHashParams
{
    static constexpr auto hashingK = 38183;
    static constexpr auto hashingDistMP = 17497;
    static constexpr auto hashingDistWP = 1;
    static constexpr auto hashingDistWPRel = 1;
};

// configure the DistributionPolicy "XMallocSIMD"
struct DistributionConfig
{
    static constexpr auto pagesize = ScatterConfig::pagesize;
};

// configure the AlignmentPolicy "Shrink"
struct AlignmentConfig
{
    static constexpr auto dataAlignment = 16;
};

// Define a new mMCator and call it ScatterAllocator
// which resembles the behaviour of ScatterAlloc
using ScatterAllocator = mallocMC::Allocator<
    mallocMC::CreationPolicies::Scatter<ScatterConfig, ScatterHashParams>,
    mallocMC::DistributionPolicies::XMallocSIMD<DistributionConfig>,
    mallocMC::OOMPolicies::ReturnNull,
    mallocMC::ReservePoolPolicies::SimpleCudaMalloc,
    mallocMC::AlignmentPolicies::Shrink<AlignmentConfig>>;

///////////////////////////////////////////////////////////////////////////////
// End of mallocMC configuration
///////////////////////////////////////////////////////////////////////////////

void run();

auto main() -> int
{
    int computeCapabilityMajor = 0;
    hipDeviceGetAttribute(
        &computeCapabilityMajor, hipDeviceAttributeComputeCapabilityMajor, 0);
    int computeCapabilityMinor = 0;
    hipDeviceGetAttribute(
        &computeCapabilityMinor, hipDeviceAttributeComputeCapabilityMinor, 0);

    if(computeCapabilityMajor < int(2))
    {
        std::cerr << "Error: Compute Capability >= 2.0 required. (is ";
        std::cerr << computeCapabilityMajor << "." << computeCapabilityMinor
                  << ")" << std::endl;
        return 1;
    }

    hipSetDevice(0);
    run();
    hipDeviceReset();

    return 0;
}

__device__ int ** arA;
__device__ int ** arB;
__device__ int ** arC;

__global__ void
createArrayPointers(int x, int y, ScatterAllocator::AllocatorHandle mMC)
{
    arA = (int **)mMC.malloc(sizeof(int *) * x * y);
    arB = (int **)mMC.malloc(sizeof(int *) * x * y);
    arC = (int **)mMC.malloc(sizeof(int *) * x * y);
}

__global__ void
fillArrays(int length, int * d, ScatterAllocator::AllocatorHandle mMC)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    arA[id] = (int *)mMC.malloc(sizeof(int) * length);
    arB[id] = (int *)mMC.malloc(sizeof(int) * length);
    arC[id] = (int *)mMC.malloc(sizeof(int) * length);

    for(int i = 0; i < length; ++i)
    {
        arA[id][i] = id * length + i;
        arB[id][i] = id * length + i;
    }
}

__global__ void addArrays(int length, int * d)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    d[id] = 0;
    for(int i = 0; i < length; ++i)
    {
        arC[id][i] = arA[id][i] + arB[id][i];
        d[id] += arC[id][i];
    }
}

__global__ void freeArrays(ScatterAllocator::AllocatorHandle mMC)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    mMC.free(arA[id]);
    mMC.free(arB[id]);
    mMC.free(arC[id]);
}

__global__ void freeArrayPointers(ScatterAllocator::AllocatorHandle mMC)
{
    mMC.free(arA);
    mMC.free(arB);
    mMC.free(arC);
}

void run()
{
    size_t block = 32;
    size_t grid = 32;
    int length = 100;
    assert((unsigned)length <= block * grid); // necessary for used algorithm

    // init the heap
    std::cerr << "initHeap...";
    ScatterAllocator mMC(
        1U * 1024U * 1024U * 1024U); // 1GB for device-side malloc
    std::cerr << "done" << std::endl;

    // device-side pointers
    int * d;
    hipMalloc((void **)&d, sizeof(int) * block * grid);

    // host-side pointers
    std::vector<int> array_sums(block * grid, 0);

    // create arrays of arrays on the device
    createArrayPointers<<<1, 1>>>(grid, block, mMC);

    // fill 2 of them all with ascending values
    fillArrays<<<grid, block>>>(length, d, mMC);

    // add the 2 arrays (vector addition within each thread)
    // and do a thread-wise reduce to d
    addArrays<<<grid, block>>>(length, d);

    hipMemcpy(
        &array_sums[0], d, sizeof(int) * block * grid, hipMemcpyDeviceToHost);

    int sum = std::accumulate(array_sums.begin(), array_sums.end(), 0);
    std::cout << "The sum of the arrays on GPU is " << sum << std::endl;

    int n = block * grid * length;
    int gaussian = n * (n - 1);
    std::cout << "The gaussian sum as comparison: " << gaussian << std::endl;

    // checking the free memory of the allocator
    if(mallocMC::Traits<ScatterAllocator>::providesAvailableSlots)
    {
        std::cout << "there are ";
        std::cout << mMC.getAvailableSlots(1024U * 1024U);
        std::cout << " Slots of size 1MB available" << std::endl;
    }

    freeArrays<<<grid, block>>>(mMC);
    freeArrayPointers<<<1, 1>>>(mMC);
    hipFree(d);
}
